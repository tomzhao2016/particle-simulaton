#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include "common2.h"
#include <set>
#include <iostream>



#define cutoff  0.01

//
//  benchmarking program
//
int main( int argc, char **argv )
{    
    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        printf( "-s <filename> to specify a summary file name\n" );
        printf( "-no turns off all correctness checks and particle output\n");
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    char *sumname = read_string( argc, argv, "-s", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    FILE *fsum = sumname ? fopen ( sumname, "a" ) : NULL;

    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    set_size( n );
    double mysize = get_size();
    init_particles( n, particles );

    /*
        assign all the particles to one of the n_row*n_col bins, my_bins is a pointer to the 
        n_row * n_col array of hash sets, each set contains the indices of particle_t objects in that bin
    */
    int n_row = (int) floor(mysize/cutoff);
    int n_col = n_row;
    std::set<int>  *my_bins = new std::set<int>[n_row*n_col];
    double xloc, yloc;
    int row, col;
    int bin_index;
    // std::cout<<"the size of the board is "<<mysize<<std::endl;
    for( int i = 0; i < n; i++ )
        {
            xloc = particles[i].x;
            yloc = particles[i].y;
            row = floor(yloc/mysize*n_row);
            col = floor(xloc/mysize*n_col);
            bin_index = row + col*n_row;
            my_bins[bin_index].insert(i);
            particles[i].bin_number = bin_index;
        }

    /*
        create an array of sets, each set contains the indices of the neighboring bins of each bin
        
    */
    std::set<int>  *bin_neighbors = new std::set<int>[n_row*n_col];
    for( int j = 0; j < n_col; j++ ){
        for( int i = 0; i < n_row; i++ ){
                 // upper left
                if (i - 1 >= 0 && j - 1 >= 0){
                    bin_neighbors[i+j*n_row].insert((i-1)+(j-1)*n_row);
                }
                // left
                if (j - 1 >= 0){
                    bin_neighbors[i+j*n_row].insert(i+(j-1)*n_row);
                }
                // lower left
                if (j - 1 >= 0 && i+1 < n_row){
                    bin_neighbors[i+j*n_row].insert(i+1+(j-1)*n_row);
                }
                // up
                if (i - 1 >= 0){
                    bin_neighbors[i+j*n_row].insert(i-1+j*n_row);
                }
                // down
                if (i + 1 < n_row){
                    bin_neighbors[i+j*n_row].insert(i+1+j*n_row);
                }
                // upper right
                if (i - 1 >= 0 && j + 1 < n_col){
                    bin_neighbors[i+j*n_row].insert((i-1)+(j+1)*n_row);
                }
                // right
                if (j + 1 < n_col){
                    bin_neighbors[i+j*n_row].insert(i+(j+1)*n_row);
                }
                // lower right
                if (i + 1 < n_row && j + 1 < n_col){
                    bin_neighbors[i+j*n_row].insert((i+1)+(j+1)*n_row);
                }

            }
        }  
    //
    //  simulate a number of time steps
    //
    double simulation_time = read_timer( );
    
    std::set<int>::iterator it2;
    std::set<int>::iterator it3;
    std::set<int>::iterator it4;

    // for( int step = 0; step < NSTEPS; step++ )
    for( int step = 0; step < NSTEPS; step++ )
    {   
        /*
            an implementation to compute forces, O(N) implementation
        */
        for (int i = 0; i < n; i++){
            particles[i].ax = particles[i].ay = 0;
            int current_bin_number = particles[i].bin_number;
            // first deal with particles in the same bin
            if(my_bins[current_bin_number].size() > 1){
                for (it2 = my_bins[current_bin_number].begin(); it2 != my_bins[current_bin_number].end(); ++it2){   
                    if(&particles[i] != &particles[*it2]){ // not the same particle
                        apply_force( particles[i], particles[*it2]); 
                    }
                }
            }
            // next deal with particels in the neighboring bins
            for (it3 = bin_neighbors[current_bin_number].begin(); it3 != bin_neighbors[current_bin_number].end(); ++it3){ 
                if(my_bins[*it3].size() > 0){
                    for (it4 = my_bins[*it3].begin(); it4 != my_bins[*it3].end(); ++it4){   
                        apply_force( particles[i], particles[*it4]);
                    }
                } 
            }

        }
        //
        //  move particles
        //
        for( int i = 0; i < n; i++ ) 
            move( particles[i] );       

          //
          //  save if necessary
          //
          if( fsave && (step%SAVEFREQ) == 0 )
              save( fsave, n, particles );
    /*
        clear the bins and reassign particles to bins
    */
    // first clear all the bins
    for (int i = 0; i < n_row*n_col; i++)
    {
        my_bins[i].clear();
    }

    // then reassign the particles
    for( int i = 0; i < n; i++ )
        {
            xloc = particles[i].x;
            yloc = particles[i].y;
            row = floor(yloc/mysize*n_row);
            col = floor(xloc/mysize*n_col);
            bin_index = row + col*n_row;
            my_bins[bin_index].insert(i);
            particles[i].bin_number = bin_index;
        }
    }

    simulation_time = read_timer( ) - simulation_time;
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time);
    //
    // Clearing space
    // 
    if(particles) 
        free( particles );
    //if(my_bins)
    //   free(my_bins);
    if( fsave )
        fclose( fsave );
    
    
    return 0;
}
