#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common2.h"
#include <set>
#include <iostream>


#define NUM_THREADS 256

extern double size;
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t * particles, int n, int max_particles_per_bin, int * my_bins, int * my_bins_count, int * bin_neighbors, int * bin_neighbors_count)
{
  // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;
    // 
    particles[tid].ax = particles[tid].ay = 0;
    int bin_number = particles[tid].bin_number;
    // particles from the same bin
    int num_par_same_bin = my_bins_count[bin_number];
    int start_index = max_particles_per_bin * bin_number;
    for(int i = 0; i < num_par_same_bin; i++){
        apply_force_gpu(particles[tid], particles[my_bins[start_index + i]]);
    }
    // particles from the neighboring bins
    int num_nb = bin_neighbors_count[bin_number];
    for(int j = 0 ; j <  num_nb; j++){
        int nb_index = bin_neighbors[8 * bin_number + j];
        int num_par_same_bin = my_bins_count[nb_index];
        int start_index = max_particles_per_bin * nb_index;
        for (int k = 0; k < num_par_same_bin; k++){
            apply_force_gpu(particles[tid], particles[my_bins[start_index + k]]);
        }
        
    }
}

__global__ void set_zero_array(int * my_bins_count, int num_bins)
{
  // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= num_bins) return;
    // 
    my_bins_count[tid] = 0;
}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

__global__ void update_bin_number (particle_t * particles, int n, double mysize, int n_row, int n_col, int * my_bins, int * my_bins_count, int max_particles_per_bin){
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;
    double xloc = particles[tid].x;
    double yloc = particles[tid].y;
    int row = (int) floor(yloc/mysize*n_row);
    int col = (int) floor(xloc/mysize*n_col);
    int bin_index = row + col*n_row;
    particles[tid].bin_number = bin_index;
    int old_count = atomicAdd(&my_bins_count[bin_index], 1);
    my_bins[max_particles_per_bin * bin_index + old_count] = particles[tid].id;
}




int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    
    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));
    
    set_size( n );
    double mysize = get_size();
    init_particles( n, particles );

    /*
        assign all the particles to one of the n_row*n_col bins, my_bins is a 2d array with fixed number of rows
        , and n_row*n_col columns, each column stores the id of particles in that bin
    */
    int n_row = (int) floor(mysize/cutoff);
    int n_col = n_row;
    int num_bins = n_col * n_row;

    // mamimum number of particles in a bin
    int max_particles_per_bin = 5 * (int) floor(n/num_bins); 

    // my_bins is a 2d array with size max_particles_per_bin by num_bins
    int  *my_bins = (int *) malloc( max_particles_per_bin * num_bins * sizeof(int) );

    // my_bins_count is a 1-d array with size 1 by num_bins
    int  *my_bins_count = (int *) malloc(num_bins * sizeof(int) );
    for(int i = 0; i < num_bins; i++){
        my_bins_count[i] = 0;
    }

    double xloc, yloc;
    int row, col;
    int bin_index;
    //std::cout << " line 167"<< std::endl;
    // assign each particle to bins, represented by my_bins and my_bins_count
    for( int i = 0; i < n; i++ )
        {
            // std::cout << "within init particles, i = "<<i << " line 131"<< std::endl;
            xloc = particles[i].x;
            yloc = particles[i].y;
            row = (int)floor(yloc/mysize*n_row);
            col = (int) floor(xloc/mysize*n_col);
            bin_index = row + col * n_row;
            particles[i].bin_number = bin_index;
            my_bins[max_particles_per_bin * bin_index + my_bins_count[bin_index]] = i;
            my_bins_count[bin_index]++;
        }
    //std::cout << " line 181"<< std::endl;
    /*
        use a 2d array of size 8 by num_bins to store the indices of neighbor bins of each bin
    */

    // bin_neighbors is a 2d array with size 8 by num_bins
    int  *bin_neighbors = (int *) malloc( 8 * num_bins * sizeof(int) );

    // bin_neighbors_count is a 1-d array with size 1 by num_bins
    int  *bin_neighbors_count = (int *) malloc(num_bins * sizeof(int) );
    for(int i = 0; i < num_bins; i++){
        bin_neighbors_count[i] = 0;
    }

    for( int j = 0; j < n_col; j++ ){
        for( int i = 0; i < n_row; i++ ){
                 // upper left
                if (i - 1 >= 0 && j - 1 >= 0){
                    bin_neighbors[(i+j*n_row) * 8 + bin_neighbors_count[i+j*n_row]] = ((i-1)+(j-1)*n_row);
                    bin_neighbors_count[i+j*n_row]++;
                }
                // left
                if (j - 1 >= 0){
                    bin_neighbors[(i+j*n_row) * 8 + bin_neighbors_count[i+j*n_row]] = (i+(j-1)*n_row);
                    bin_neighbors_count[i+j*n_row]++;
                }
                // lower left
                if (j - 1 >= 0 && i+1 < n_row){
                    bin_neighbors[(i+j*n_row) * 8 + bin_neighbors_count[i+j*n_row]] = (i+1+(j-1)*n_row);
                    bin_neighbors_count[i+j*n_row]++;
                }
                // up
                if (i - 1 >= 0){
                    bin_neighbors[(i+j*n_row) * 8 + bin_neighbors_count[i+j*n_row]] = (i-1+j*n_row);
                    bin_neighbors_count[i+j*n_row]++;
                }
                // down
                if (i + 1 < n_row){
                    bin_neighbors[(i+j*n_row) * 8 + bin_neighbors_count[i+j*n_row]] = (i+1+j*n_row);
                    bin_neighbors_count[i+j*n_row]++;
                }
                // upper right
                if (i - 1 >= 0 && j + 1 < n_col){
                    bin_neighbors[(i+j*n_row) * 8 + bin_neighbors_count[i+j*n_row]] = ((i-1)+(j+1)*n_row);
                    bin_neighbors_count[i+j*n_row]++;
                }
                // right
                if (j + 1 < n_col){
                    bin_neighbors[(i+j*n_row) * 8 + bin_neighbors_count[i+j*n_row]] = (i+(j+1)*n_row);
                    bin_neighbors_count[i+j*n_row]++;
                }
                // lower right
                if (i + 1 < n_row && j + 1 < n_col){
                    bin_neighbors[(i+j*n_row) * 8 + bin_neighbors_count[i+j*n_row]] = ((i+1)+(j+1)*n_row);
                    bin_neighbors_count[i+j*n_row]++;
                }

            }
        } 
    //    std::cout << " line 240"<< std::endl;

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    // copy my_bins to GPU
    int *d_my_bins;
    hipMalloc((void **) &d_my_bins, max_particles_per_bin * num_bins * sizeof(int));

    // copy my_bins_count to GPU
    int *d_my_bins_count;
    hipMalloc((void **) &d_my_bins_count, num_bins * sizeof(int));

    // copy bin_neighbors to GPU, bin_neighbors need not update
    int *d_bin_neighbors;
    hipMalloc((void **) &d_bin_neighbors, 8 * num_bins * sizeof(int));

    // copy bin_neighbors_count to GPU, bin_neighbors_count need not update
    int *d_bin_neighbors_count;
    hipMalloc((void **) &d_bin_neighbors_count, num_bins * sizeof(int));
    //
    hipMemcpy(d_my_bins, my_bins, max_particles_per_bin * num_bins * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    //
    hipMemcpy(d_my_bins_count, my_bins_count, num_bins * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    //
    hipMemcpy(d_bin_neighbors, bin_neighbors, 8 * num_bins * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    //
    hipMemcpy(d_bin_neighbors_count, bin_neighbors_count, num_bins * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    //std::cout << " line 278"<< std::endl;
    //
    //  simulate a number of time steps
    //
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //
        //double  before_compute_forces = read_timer();
        //std::cout << "step "<< step<< " line 290"<< std::endl;
	    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
	    compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n, max_particles_per_bin, d_my_bins, d_my_bins_count, d_bin_neighbors, d_bin_neighbors_count);
        //double  after_compute_forces = read_timer();
        //std::cout << "compute_forces time " << after_compute_forces - before_compute_forces << std::endl;
        //std::cout << "step "<< step<< " line 295"<< std::endl;
        //
        //  move particles
        //
	    move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        //std::cout << "step "<< step<< " line 300"<< std::endl;
        // reset d_my_bins_count
        set_zero_array<<< blks, NUM_THREADS >>> (d_my_bins_count, num_bins);
        //std::cout << "step "<< step<< " line 305"<< std::endl;
        // update particles' bin_number 
        update_bin_number <<< blks, NUM_THREADS >>> (d_particles, n, mysize, n_row, n_col, d_my_bins, d_my_bins_count, max_particles_per_bin);
        //std::cout << "step "<< step<< " line 308"<< std::endl;
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
        // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
